#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hiprand/hiprand.h>


#if 0
#define dfloat double
#define dfloatString "double"
#else
#define dfloat float
#define dfloatString "float"
#endif

#define p_dim 3
#define p_Nfields 4

// scraped from recent 


#define p_Nvgeo 9

#define p_G00ID 0
#define p_G01ID 1
#define p_G02ID 2
#define p_G11ID 3
#define p_G12ID 4
#define p_G22ID 5
#define p_GWJID 6


void generateRandArray(int sz, dfloat * a){
  //  a = (dfloat*) calloc(sz, sizeof(dfloat)); 
  for (int n=0; n<sz;++n){
    a[n] = drand48()-0.5;
    //printf("a[%d] = %f \n ", n, a[n]);
  }
}

//coalested reads
__global__ void geofactorsKernelv1(const int Nelements, 
				   const dfloat * __restrict__ vgeo,
				   const dfloat * __restrict__ Dq,
				   dfloat * __restrict__ Aq  ){

  int p_Np = blockDim.x; // define this
  int e  = blockIdx.x; 
  const int t  = threadIdx.x;

  const dfloat rx = vgeo[e*p_Nvgeo + 0];
  const dfloat sx = vgeo[e*p_Nvgeo + 1];
  const dfloat tx = vgeo[e*p_Nvgeo + 2];
  const dfloat ry = vgeo[e*p_Nvgeo + 3];
  const dfloat sy = vgeo[e*p_Nvgeo + 4];
  const dfloat ty = vgeo[e*p_Nvgeo + 5];
  const dfloat rz = vgeo[e*p_Nvgeo + 6];
  const dfloat sz = vgeo[e*p_Nvgeo + 7];
  const dfloat tz = vgeo[e*p_Nvgeo + 8];

  const int Dbase = e*p_dim*p_Np*p_Nfields + t;
  
  dfloat ur = Dq[Dbase + p_Np*p_dim*0 + p_Np*0];
  dfloat us = Dq[Dbase + p_Np*p_dim*0 + p_Np*1];
  dfloat ut = Dq[Dbase + p_Np*p_dim*0 + p_Np*2];
  dfloat vr = Dq[Dbase + p_Np*p_dim*1 + p_Np*0];
  dfloat vs = Dq[Dbase + p_Np*p_dim*1 + p_Np*1];
  dfloat vt = Dq[Dbase + p_Np*p_dim*1 + p_Np*2];
  dfloat wr = Dq[Dbase + p_Np*p_dim*2 + p_Np*0];
  dfloat ws = Dq[Dbase + p_Np*p_dim*2 + p_Np*1];
  dfloat wt = Dq[Dbase + p_Np*p_dim*2 + p_Np*2];
  dfloat pr = Dq[Dbase + p_Np*p_dim*3 + p_Np*0];
  dfloat ps = Dq[Dbase + p_Np*p_dim*3 + p_Np*1];
  dfloat pt = Dq[Dbase + p_Np*p_dim*3 + p_Np*2];

  const int base = e*p_Np*p_Nfields + t;

  Aq[base + p_Np*0] = rx*pr + sx*ps + tx*pt;
  Aq[base + p_Np*1] = ry*pr + sy*ps + ty*pt;
  Aq[base + p_Np*2] = rz*pr + sz*ps + tz*pt;
  

  dfloat divU = rx*ur + sx*us + tx*ut;
  divU += ry*vr + sy*vs + ty*vt;
  divU += rz*wr + sz*ws + tz*wt;

  Aq[base + p_Np*3] = divU;
}

void gpuFillRand(int N, dfloat **h_v, dfloat **c_v){

  *h_v = (dfloat*) calloc(N, sizeof(dfloat));

  for(int n=0;n<N;++n) h_v[0][n] = drand48();

  hipMalloc(c_v, N*sizeof(dfloat));

  if(c_v==NULL){
    printf("gpuFillRand(%d,..,..) failed\n", N);
  }
  
  hipMemcpy(*c_v, *h_v, N*sizeof(dfloat), hipMemcpyHostToDevice);


}

void gpuBlasGemm(hipblasHandle_t &handle, const dfloat *A, const dfloat *B, 
		 dfloat *C, const int m, const int n, const int k, const int option) {
 
  if (option == 1){
    //tranpose the matrices
    //7* p_Np, E,  p_Np
    //m, n, k

    //m: number of rows of matrix op(A) -- first matrix --  and C.
    // op(B) = trans(q) = trans(p_Np*E) - > E
    //n: number of columns in op(b) -- second matrix - and C 7*p_Np
    // op(A) = trans(D) = trans(7*p_Np x p_Np) -> [p_Np x 7*p_Np] -> 7p_Np
    // k the remaining number which is p_Np
    // result is C^T so size [7*Np x E] 
    // int lda=m,ldb=k,ldc=m;
    //C [p_Np*7 x E]
    //NW int lda=m,ldb=n,ldc=k;
    int lda = m, ldb =k, ldc=n; 
    //NW int lda = k, ldb = m, ldc =n;
    //NW int lda = k, ldb =n, ldc = m;
    //NW int lda = n, ldb =k, ldc =m;
    //NW int lda =n, ldb =m, ldc = k;



    const dfloat alf = 1;
    const dfloat bet = 0;
    const dfloat *alpha = &alf;
    const dfloat *beta = &bet;


    // Do the actual multiplication
    if(sizeof(dfloat)==8)
      hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, m, k, 
		  (double*)alpha, (double*)B, lda, 
		  (double*)A, lda, 
		  (double*)beta, 
		  (double*)C, ldc);
    else
      hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, m, k, (float*)alpha, (float*)B, ldb, (float*)A, lda, (float*)beta, (float*)C, ldc);
  }
  else{
    //default

    int lda=m,ldb=k,ldc=m;
    const dfloat alf = 1;
    const dfloat bet = 0;
    const dfloat *alpha = &alf;
    const dfloat *beta = &bet;


    // Do the actual multiplication
    if(sizeof(dfloat)==8)
      hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, 
		  (double*)alpha, (double*)A, lda, 
		  (double*)B, ldb, 
		  (double*)beta, 
		  (double*)C, ldc);
    else
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, (float*)alpha, (float*)A, lda, (float*)B, ldb, (float*)beta, (float*)C, ldc);
  }

}



int main(int argc, char **argv){

  size_t aux;
  printf("size of size_t %lud \n", sizeof(aux));
  int E = (argc>=2) ? atoi(argv[1]):512;
  int p_N = (argc>=3) ? atoi(argv[2]):5;
  //int option = (argc>=4) ? atoi(argv[3]):1;  
  //int p_Ne = (argc>=5) ? atoi(argv[4]):1;
  //int p_Nb = (argc>=6) ? atoi(argv[5]):1;

  int p_Np = ((p_N+1)*(p_N+2)*(p_N+3))/6;

  printf("E= %d p_N = %d p_Np = %d \n", E, p_N, p_Np);

  // number of geometric factors
  
  int Niter = 10;
  //(2*p_Np-1)*7*p_Np + 14*p_Np;
  unsigned long long int gflops= p_dim*p_Nfields*p_Np*p_Np*2 + p_Np*2*15;

  dfloat  *h_D, *h_q, *h_Dq, *h_Aq, *h_vgeo;
  dfloat  *d_D, *d_q, *d_Dq, *d_Aq, *d_vgeo;

  srand48(12345);  

  // allocate D
  gpuFillRand( (p_dim*p_Np)*p_Np, &h_D, &d_D); 

  //allocate q
  gpuFillRand( p_Nfields*p_Np*E,   &h_q,    &d_q); 

  //allocate Dq
  gpuFillRand(p_Nfields*p_dim*p_Np*E,   &h_Dq,    &d_Dq);
  
  //allocate Aq
  gpuFillRand(p_Nfields*p_Np*E,   &h_Aq,    &d_Aq); 

  //allocate geofactors
  gpuFillRand(p_Nvgeo*E,   &h_vgeo,    &d_vgeo);


  size_t free, total;

  printf("\n");

  hipMemGetInfo(&free,&total); 

  printf("before %17.18lu B free of total %17.18lu B\n",free,total);



  //3) use cublasDgemm -> outputs 7k*p_NpxNel array

  // Create a handle for CUBLA
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // create events
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);


  hipEventRecord(start);

  for(int it=0;it<Niter;++it){
    //  [Dr;Ds;Dt] x [r,u,v,w,p]
    gpuBlasGemm(handle, d_D, d_q, d_Dq, p_dim*p_Np, p_Nfields*E,  p_Np, 0);

    //printf("done! %d \n", it);
    geofactorsKernelv1<<< E, p_Np >>> (E, d_vgeo, d_Dq, d_Aq);
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float  elapsedCublas =0.0f;

  hipEventElapsedTime(&elapsedCublas, start, stop);
  elapsedCublas /= (1000.*Niter);

  printf("[ DDD %d  %17.15f %17.15f ]\n", p_Np*E,  gflops*E/(elapsedCublas*1.e9), p_Np*E/(elapsedCublas*1.e9)); 

  //full version
  printf("TIME %5.4e flops = %llu  GFLOPS: %17.17f \n",elapsedCublas,gflops, (((dfloat)gflops*(dfloat)E/10e8)/elapsedCublas));

  hipMemcpy(h_Aq, d_Aq, p_Nfields*E*p_Np*sizeof(dfloat), hipMemcpyDeviceToHost);
  hipMemGetInfo(&free,&total); 

  printf(" after %17.18lu  B free of total %17.18lu B\n",free,total);

  dfloat normAq = 0;

  for(int n=0;n<E*p_Np*p_Nfields;++n)
    normAq += (h_Aq[n]*h_Aq[n]);
  normAq = sqrt(normAq);

  printf("CUDA-CUBLAS: error Aq = %17.15lf\n", normAq);

  // Destroy the handle
  hipblasDestroy(handle);

  exit(0);
  return 0;

}

